#include "hip/hip_runtime.h"
#include "ADMM.h"
#include "HexEle.h"

#include "ElementMesh.hpp"
#include "Element.hpp"
#include "vecmath.h"
#include "Util.hpp"
#include <time.h>  
#include <fstream>
ADMMStepper::ADMMStepper():NSteps(100),nThread(256),ro(10),tol(0.0001f),
  maxDist(0.15f),roMult(1.5f),outname("out.txt")
{}

ADMMStepper::~ADMMStepper()
{
  ///@TODO delete arrays and cuda free
}

float3 vec2float(const Vector3f & v)
{
  return make_float3(v[0],v[1],v[2]);
}

Vector3f float2vec(const float3 & v)
{
  return Vector3f(v.x, v.y, v.z);
}

void ADMMStepper::setZdev(const std::vector<Vector3f> & vec)
{
  int nEle = (int)m->e.size();
  for(int ii = 0;ii<nEle;ii++){
    ADMMInfo * admm = &(hostadmm[ii]);
    for(int jj = 0;jj<NVERT;jj++){
      int vIdx = m->e[ii]->at(jj);
      admm->Z[jj] = vec2float(vec[vIdx]);
    }
  }
  hipMemcpy(devadmm, hostadmm, nEle*sizeof(ADMMInfo), hipMemcpyHostToDevice);
}

float ADMMStepper::getEnergy()
{
  int nEle = (int)m->e.size();
  int nBlock = nEle/nThread + ((nEle % nThread)!=0);
  GetEnergy<<<nBlock, nThread>>>(devXX, devadmm, Edev);
  hipMemcpy(Ehost, Edev, nEle*sizeof(float), hipMemcpyDeviceToHost);
  float ene = 0;
  for(int ii= 0;ii<nEle;ii++){
    ene += Ehost[ii];
  }
  for(unsigned int ii = 0;ii<m->fe.size();ii++){
    ene -= Vector3f::dot(m->fe[ii], Z[ii]);
  }
  return ene;
}

//get forces for the mesh excluding ADMM terms
void ADMMStepper::getForce(std::vector<Vector3f> & ff)
{
  int nEle = (int)m->e.size();
  int nBlock = nEle/nThread + ((nEle % nThread)!=0);
  
  GetIntForce<<<nBlock, nThread>>>(devXX, devadmm, fdev);
  hipMemcpy(fhost, fdev, nEle*NVERT*sizeof(float3), hipMemcpyDeviceToHost);
  //cumulate internal forces
  ff.assign(ff.size(), Vector3f::ZERO);
  for(int ii= 0;ii<nEle;ii++){
    for(int jj = 0;jj<NVERT;jj++){
      int vidx = m->e[ii]->at(jj);
      float3 ftmp = fhost[NVERT*ii + jj];
      Vector3f fint(ftmp.x,ftmp.y,ftmp.z);
      ff[vidx] += fint;
    }
  }
  for(int ii = 0;ii<m->fe.size();ii++){
    ff[ii] += m->fe[ii];
  }
  for(unsigned int ii = 0;ii<m->fixed.size();ii++){
    if (m->fixed[ii]){
      ff[ii] = Vector3f::ZERO;
    }
  }
}

void ADMMStepper::stepGrad()
{
  int nEle = (int)m->e.size();
  
  //change in Z
  std::vector<Vector3f> dZ(Z.size());
  //internal + external forces
  std::vector<Vector3f> ff(Z.size());
  float hh = 0.001f;
  for(int iter = 0;iter<NSteps;iter++){
    std::cout<<iter<<"\n";
    setZdev(Z);
    float ene = getEnergy();
    getForce(ff);
    float mag = 0;
    for(unsigned int ii = 0;ii<ff.size(); ii++){
      for(int jj = 0;jj<3;jj++){
        mag += std::abs(ff[ii][jj]);
      }
    }
    mag = mag/ff.size();
    if(mag<tol){
      break;
    }
    float ene1=ene;
    while(1){
      dZ = hh * ff;
      Z = m->x + dZ;
      setZdev(Z);
      ene1 = getEnergy();
      if(ene1<ene){
        hh=hh*1.5f;
        break;
      }else{
        hh=hh/2;
        if(hh<1e-15f){
          break;
        }
      }
    }
    std::cout<<hh<<" "<<ene1<<"\n";
    
    m->x = Z;
  }
}

int
ADMMStepper::oneStep()
{
  int nEle = (int)m->e.size();
  int nBlock = nEle/nThread + ((nEle % nThread)!=0);
  
  
  //change in Z
  std::vector<Vector3f> dZ(Z.size());
  //internal + external forces
  std::vector<Vector3f> ff(Z.size());

  setZdev(Z);
  float ene = getEnergy();
  getForce(ff);
  float eleSize = m->eleSize();
  float maxRo = 0;
  float maxDiff = 0;
  //adjust ro
  for (int ii = 0; ii < nEle; ii++){
    ADMMInfo * admm = &(hostadmm[ii]);
    for (int jj = 0; jj < NVERT; jj++){
      int vidx = m->e[ii]->at(jj);
      Vector3f zz = Z[vidx];
      Vector3f xx = float2vec(hostxx[ii*NVERT + jj]);
      float diff = (xx - zz).abs();
      if (diff > maxDiff){
        maxDiff = diff;
      }
      if (admm->ro > maxRo){
        maxRo = admm->ro;
      }
      if (diff > maxDist*eleSize){
        admm->ro *= roMult;
        break;
      }
    }
  }
  std::cout << "maxro " << maxRo << "\n";
  std::cout << "maxdiff " << maxDiff << "\n";
  for (int ii = 0; ii < nEle; ii++){
    //copy Z and y to device
    ADMMInfo * admm = &(hostadmm[ii]);
    for (int jj = 0; jj < NVERT; jj++){
      int vIdx = m->e[ii]->at(jj);
      admm->Z[jj] = vec2float(Z[vIdx]);
      admm->y[jj] = vec2float(l[ii][jj]);
    }
  }
  hipMemcpy(devadmm, hostadmm, nEle*sizeof(ADMMInfo), hipMemcpyHostToDevice);
  admmMinEleDup <<<nBlock, nThread >>>(devXX, devxx, devadmm);
  hipMemcpy(hostxx, devxx, nEle*NVERT*sizeof(float3), hipMemcpyDeviceToHost);

  //update z closed form
  for (size_t ii = 0; ii < m->X.size(); ii++){
    Z[ii] = Vector3f::ZERO;
  }
  //add per element variables and multipliers
  for (int ii = 0; ii < nEle; ii++){
    Element * ele = m->e[ii];
    for (int jj = 0; jj < ele->nV(); jj++){
      Vector3f xx = float2vec(hostxx[ii*NVERT + jj]);
      Z[ele->at(jj)] += xx + l[ii][jj] / ro;
    }
  }

  //add force variables
  for (size_t ii = 0; ii < m->fe.size(); ii++){
    Z[ii] += (1.0f / ro)*m->fe[ii];
  }

  //divide
  for (size_t ii = 0; ii < m->x.size(); ii++){
    Z[ii] /= (float)N[ii];
  }

  //fix constrained vertices
  for (auto ii = 0; ii < m->fixed.size(); ii++){
    if (m->fixed[ii]){
      Z[ii] = m->x[ii];
    }
  }

  //update multiplier for elements
  for (size_t ii = 0; ii < nEle; ii++){
    Element * ele = m->e[ii];
    for (int jj = 0; jj < ele->nV(); jj++){
      Vector3f xx = float2vec(hostxx[ii*NVERT + jj]);
      l[ii][jj] += ro*(xx - Z[ele->at(jj)]);
    }
  }

  float hh = 1;
  float mag = 0;
  for (unsigned int ii = 0; ii < ff.size(); ii++){
    for (int jj = 0; jj < 3; jj++){
      mag += std::abs(ff[ii][jj]);
    }
  }
  mag = mag / ff.size();
  if (mag < tol){
    return -1;
  }

  float ene1 = ene;
  for (unsigned int ii = 0; ii < Z.size(); ii++){
    dZ[ii] = Z[ii] - m->x[ii];
  }
  //while (1){
  //  Z = m->x + hh*dZ;
  //  setZdev(Z);
  //  ene1 = getEnergy();
  //  if (ene1 < ene){
  //    break;
  //  }
  //  else{
  //    hh = hh / 2;
  //    if (hh < 1e-15){
  //      break;
  //    }
  //  }
  //}
  std::cout << hh << " " << ene1 << "\n";
  m->x = Z;
  return 0;
}

void ADMMStepper::init(ElementMesh * _m)
{
  m = _m;
  prevE = m->getEnergy();
  initVar();
}

void
ADMMStepper::initVar()
{
  ElementMesh * e = m;
  int nEle = (int)e->e.size();
  l.resize(e->e.size());
  N.assign(e->X.size(),0);
  for(size_t ii= 0; ii<e->e.size();ii++){
    Element * ele = e->e[ii];
    l[ii].resize(ele->nV(), Vector3f::ZERO);
    for(int jj = 0;jj<ele->nV();jj++){
      N[ele->at(jj)] ++;
    }
  }

  Z = e->x;
  int xsize = NVERT * nEle;
  hostXX = new float3[NVERT];
  hostxx = new float3 [xsize];
  Element * ele = e->e[0];
  
  for(int ii = 0;ii<NVERT;ii++){
    hostXX[ii].x= e->X[ele->at(ii)][0];
	  hostXX[ii].y= e->X[ele->at(ii)][1];
	  hostXX[ii].z= e->X[ele->at(ii)][2];
  }
  hostadmm = new ADMMInfo[nEle];
  hipMalloc((void**)&devXX,        NVERT*sizeof(float3));
  hipMalloc((void**)&devxx,   nEle*NVERT*sizeof(float3));
  hipMalloc((void**)&devadmm, nEle*sizeof(ADMMInfo));
  hipMemcpy(devXX,hostXX, NVERT*sizeof(float3),hipMemcpyHostToDevice);

  int xIdx = 0;
  for(size_t ii = 0;ii<e->e.size();ii++){
    Element * ele = e->e[ii];
    hostadmm[ii].ro = ro;
    for(int jj = 0;jj<ele->nV();jj++){
      hostxx[xIdx] = vec2float(e->x[ele->at(jj)]);
      xIdx++;
    }
  }
  hipMemcpy(devxx, hostxx, xsize * sizeof(float3), hipMemcpyHostToDevice);
  initHexEle();

  Ehost = new float[nEle];
  hipMalloc(&Edev,nEle * sizeof(float));
  fhost = new float3[nEle*NVERT];
  hipMalloc(&fdev, nEle*NVERT * sizeof(float3));

}
